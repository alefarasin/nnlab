#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel CUDA che viene eseguito sulla GPU
__global__ void helloKernel() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hello World dal thread %d!\n", idx);
}

int main() {
    printf("Hello World dalla CPU (host)!\n");
    
    // Configurazione per il lancio del kernel
    int numThreads = 8;
    int numBlocks = 2;
    
    // Lancio del kernel sulla GPU
    helloKernel<<<numBlocks, numThreads>>>();
    
    // Sincronizzazione: aspetta che tutti i thread GPU finiscano
    hipDeviceSynchronize();
    
    // Controlla eventuali errori CUDA
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Errore CUDA: %s\n", hipGetErrorString(error));
        return -1;
    }
    
    printf("Programma completato!\n");
    return 0;
}